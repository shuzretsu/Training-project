// Matrix Multiplication CUDA

#include <hip/hip_runtime.h>

extern "C" void matrix_vector_mult_cuda(float* A, float* x, float* b, int n);

__global__ void matvec_kernel(float* A, float* x, float* b, int n) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < n) {
        float sum = 0.0f;
        for (int j = 0; j < n; ++j) {
            sum += A[row * n + j] * x[j];
        }
        b[row] = sum;
    }
}

extern "C" void matrix_vector_mult_cuda(float* A, float* x, float* b, int n) {
    float *d_A, *d_x, *d_b;
    
    // Allocate memory on the device (GPU)
    hipMalloc((void**)&d_A, n * n * sizeof(float));
    hipMalloc((void**)&d_x, n * sizeof(float));
    hipMalloc((void**)&d_b, n * sizeof(float));

    // Copy data from host (CPU) to device (GPU)
    hipMemcpy(d_A, A, n * n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_x, x, n * sizeof(float), hipMemcpyHostToDevice);

    // Launch the kernel with n threads whic one for each row of matrix A
    int blockSize = 256; // we can adjust thee block size according to your GPU architecture(search on google about that)
    int gridSize = (n + blockSize - 1) / blockSize;
    matvec_kernel<<<gridSize, blockSize>>>(d_A, d_x, d_b, n);

    // Copy result from (GPU) to host (CPU)
    hipMemcpy(b, d_b, n * sizeof(float), hipMemcpyDeviceToHost);

    // Free the device memory
    hipFree(d_A);
    hipFree(d_x);
    hipFree(d_b);
}
